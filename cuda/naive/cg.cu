#include "hip/hip_runtime.h"
#include<hipblas.h>

extern "C" {
    #include "cg.h"
}

const double TOLERANCE = 1.0e-10;

__device__ double atomic_Add(double* address, double val){
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                        __longlong_as_double(assumed)));
    } while (assumed != old);

    return __longlong_as_double(old);
}

__global__ void kernel_mv(const double * const A, const double * const X, double * const AA, int m) {
    int i,j,k,l;
    int t = blockDim.x*gridDim.x;
    int id = threadIdx.x + blockIdx.x*blockDim.x;
    double temp;

    for (i=0;i<(m*m/t);i++){
        j=(id+i*t);
        k=(id+i*t)/m;
	l=(id+i*t)%m;
	if (j<m*m){
            temp=A[j]*X[k];
            atomic_Add(&AA[l], temp); 
	}
    }
}

__global__ void kernel_zero(double *AA, int m) {
    int i,j;
    int t = blockDim.x*gridDim.x;
    int id = threadIdx.x + blockIdx.x*blockDim.x;

    for (i=0;i<m/t;i++){
        j=id+i*t;
        AA[j]=0.0;
    }
}

void cgsolver( double *A, double *b, double *x, int m, int n ){
        double * d_r;
        double * d_rt;
        double * d_p;
        double * d_pt;
        double * d_Ap;
        double * d_tmp;

        double alpha, temp;
	double rsold, rsnew;
        int incx = 1;
        int incy = 1;
        int lda = m;
        double al = 1.;
        double be = 0.;

        int k = 0;

        double *d_A, *d_x, *d_b;

	hipMalloc((void**)&d_r  ,sizeof(double)*n);
        hipMalloc((void**)&d_rt ,sizeof(double)*n);
        hipMalloc((void**)&d_p  ,sizeof(double)*n);
        hipMalloc((void**)&d_pt ,sizeof(double)*n);
        hipMalloc((void**)&d_Ap ,sizeof(double)*n);
        hipMalloc((void**)&d_tmp,sizeof(double)*n);
	hipMalloc((void**)&d_b  ,sizeof(double)*n);
        hipMalloc((void**)&d_A  ,sizeof(double)*n*m);
        hipMalloc((void**)&d_x  ,sizeof(double)*n);

        hipMemcpy(d_A, A, sizeof(double)*n*m, hipMemcpyHostToDevice);
        hipMemcpy(d_x, x, sizeof(double)*n,   hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(double)*n,   hipMemcpyHostToDevice);

        hipblasHandle_t cublasHandle = 0;
        hipblasCreate(&cublasHandle);

	kernel_mv<<<10000,1000>>>(d_A, d_x, d_Ap, m); hipDeviceSynchronize();
	temp=-1.0;

	hipblasDcopy(cublasHandle, n, d_b, 1, d_tmp, 1); hipDeviceSynchronize();
        hipblasDaxpy(cublasHandle, n, &temp, d_Ap, 1, d_tmp, 1); hipDeviceSynchronize();
        hipblasDcopy(cublasHandle, n, d_tmp, 1, d_r, 1); hipDeviceSynchronize();
        hipblasDcopy(cublasHandle, n, d_r, 1, d_p, 1); hipDeviceSynchronize();
        hipblasDcopy(cublasHandle, n, d_r, 1, d_rt, 1); hipDeviceSynchronize();
        hipblasDcopy(cublasHandle, n, d_p, 1, d_pt, 1); hipDeviceSynchronize();

	hipblasDdot(cublasHandle, n, d_r, incx, d_rt, incy, &rsold); hipDeviceSynchronize();

        while ( k < n ){
		kernel_zero<<<10,1000>>>(d_Ap, m); hipDeviceSynchronize();
		kernel_mv<<<10000,1000>>>(d_A, d_p, d_Ap, m); hipDeviceSynchronize();
		
		hipblasDdot(cublasHandle, n, d_pt, incx, d_Ap, incy, &temp); hipDeviceSynchronize();
		alpha=rsold/fmax(temp, NEARZERO);

                hipblasDaxpy(cublasHandle, n, &alpha, d_p, 1, d_x, 1); hipDeviceSynchronize();
		temp = -alpha;
                hipblasDaxpy(cublasHandle, n, &temp, d_Ap, 1, d_r, 1); hipDeviceSynchronize();
                hipblasDdot(cublasHandle, n, d_r, incx, d_r, incy, &rsnew); hipDeviceSynchronize();

                if ( sqrt(rsnew) < TOLERANCE ) break;

                hipblasDcopy(cublasHandle, n, d_r, 1, d_tmp, 1); hipDeviceSynchronize();
		temp = rsnew/rsold;
                hipblasDaxpy(cublasHandle, n, &temp, d_p, 1, d_tmp, 1); hipDeviceSynchronize();
                hipblasDcopy(cublasHandle, n, d_tmp, 1, d_p, 1); hipDeviceSynchronize();
                hipblasDcopy(cublasHandle, n, d_p, 1, d_pt, 1); hipDeviceSynchronize();
		rsold = rsnew;

                k++;
        }

        printf("\t[STEP %d] residual = %E\n",k,sqrt(rsold));

	hipFree(d_A);
	hipFree(d_x);
        hipFree(d_r);
        hipFree(d_rt);
        hipFree(d_p);
        hipFree(d_pt);
        hipFree(d_Ap);
        hipFree(d_tmp);
	hipblasDestroy(cublasHandle);
}

/*
Sparse version of the cg solver
*/

extern "C" void cgsolver_sparse( double *Aval, int *Irn, int *Jcn, double *b, double *x, int n){
	double * r;
	double * rt;
	double * p;
	double * pt;
	double rsold;
	double rsnew;
	double * Ap;
	double * tmp;
	double alpha;

	int incx = 1;
	int incy = 1;

	int k = 0;

	r = (double*) malloc(n* sizeof(double));
	rt = (double*) malloc(n* sizeof(double));
	p = (double*) malloc(n* sizeof(double));
	pt = (double*) malloc(n* sizeof(double));
	Ap = (double*) malloc(n* sizeof(double));
	tmp = (double*) malloc(n* sizeof(double));

//    r = b - A * x;
	smvm(n, Aval, Jcn, Irn, x, Ap);
	cblas_dcopy(n,b,1,tmp,1);	
	cblas_daxpy(n, -1.  , Ap,1, tmp, 1 );
	cblas_dcopy(n,tmp,1,r,1);

//    p = r;
	cblas_dcopy (n, r, incx, p, incy);

	cblas_dcopy(n,r,1,rt,1);	
	cblas_dcopy(n,p,1,pt,1);	
//    rsold = r' * r;
	rsold = cblas_ddot (n,r,incx,rt,incy);



//    for i = 1:length(b)
	while ( k < n ){
//        Ap = A * p;
		smvm(n, Aval, Jcn, Irn, p, Ap);
//        alpha = rsold / (p' * Ap);
		alpha = rsold / fmax( cblas_ddot(n, pt, incx, Ap, incy ), NEARZERO );
//        x = x + alpha * p;
		memset(x, 0, n*sizeof(double));
		cblas_daxpy(n, alpha, p, 1, x, 1);
//        r = r - alpha * Ap;
		cblas_daxpy(n, -alpha, Ap, 1, r, 1);
//        rsnew = r' * r;
		rsnew = cblas_ddot (n,r,incx,r,incy);
//        if sqrt(rsnew) < 1e-10
//              break;
		if ( sqrt(rsnew) < TOLERANCE ) break;             // Convergence test
//        p = r + (rsnew / rsold) * p;
		memset(p, 0, n*sizeof(double));
		cblas_dcopy(n,r,1,tmp,1);	
		cblas_daxpy(n, (double)(rsnew/rsold), p, 1, tmp, 1);
		cblas_dcopy(n,tmp,1,p,1);
	
		cblas_dcopy(n,p,1,pt,1);	
//        rsold = rsnew;
		rsold = rsnew;
		k++;
	}

	printf("\t[STEP %d] residual = %E\n",k,sqrt(rsold));

	free(r);
	free(rt);
	free(p);
	free(pt);
	free(Ap);
	free(tmp);
}

/*
Sparse matrix vector multiplication
*/

extern "C" void smvm(int m, const double* val, const int* col, const int* row, const double* x, double* y)
{
	for (int i=0; i<m; ++i) {
		y[i] = 0.0;
		for (int j=row[i]; j<row[i+1]; ++j){
			y[i] += val[j-1]*x[col[j-1]-1];
		}
	}
}




/*
Initialization of the source term b 
*/

extern "C" double * init_source_term(int n, double h){
	double * f;
	int i;
	f  = (double*) malloc(n*sizeof(double*));

	for(i = 0; i < n; i++) {
		f[i] = (double)i * -2. * M_PI * M_PI * sin(10.*M_PI*i*h) * sin(10.*M_PI*i*h);
	}
	return f;
}
